#include <stdio.h>
#include<hip/hip_runtime.h>

#include<time.h>
#include<stdlib.h>

#define BLOCK_NUM 32   //块数量
#define THREAD_NUM 32 // 每个块中的线程数
#define R_SIZE 1024//BLOCK_NUM * THREAD_NUM
#define M_SIZE R_SIZE * R_SIZE

__global__ void mat_mul(int *mat1, int *mat2, int *result) {
    const int bid = blockIdx.x;
    const int tid = threadIdx.x;
    // 每个线程计算一行
    const int row = bid * THREAD_NUM + tid;
    for (int c = 0; c < R_SIZE; c++) {
        for (int n = 0; n < R_SIZE; n++) {
            result[row*R_SIZE+c] += mat1[row*R_SIZE+n] * mat2[n*R_SIZE+c];
        }
    }
}

int main(int argc, char *argv[]) {
    int *mat1, *mat2, *result;
    int *g_mat1, *g_mat2, *g_mat_result;
    
    // 用一位数组表示二维矩阵
    mat1 = (int*) malloc(M_SIZE * sizeof(int));
    mat2 = (int*) malloc(M_SIZE * sizeof(int));
    result = (int*) malloc(M_SIZE * sizeof(int));

    // initialize
    for (int i = 0; i < M_SIZE; i++) {
        mat1[i] = rand()/1000000;
        mat2[i] = rand()/1000000;
        result[i] = 0;
        
    }
    clock_t start, finish;    
    start = clock();
    hipMalloc((void **)&g_mat1, sizeof(int) * M_SIZE);
    hipMalloc((void **)&g_mat2, sizeof(int) * M_SIZE);
    hipMalloc((void **)&g_mat_result, sizeof(int) * M_SIZE);

    hipMemcpy(g_mat1, mat1, sizeof(int) * M_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(g_mat2, mat2, sizeof(int) * M_SIZE, hipMemcpyHostToDevice);

    mat_mul<<<BLOCK_NUM, THREAD_NUM>>>(g_mat1, g_mat2, g_mat_result);

    hipMemcpy(result, g_mat_result, sizeof(int) * M_SIZE, hipMemcpyDeviceToHost);
    finish = clock();
    printf("total times: %.3f\n",(double)(finish-start)/CLOCKS_PER_SEC);
     for(int i=0;i<10;i++)
       printf("%d ",result[i]);
    printf("\n");
    hipFree(g_mat1);hipFree(g_mat2);hipFree(g_mat_result);
    free(mat1); free(mat2); free(result);
    return 0;
}
